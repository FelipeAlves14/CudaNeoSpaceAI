#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>

__global__ void calculaCelulas(float *matrizA, float *matrizB, float *matrizC, int N)
{
    int linha = blockIdx.y * blockDim.y + threadIdx.y;
    int coluna = blockIdx.x * blockDim.x + threadIdx.x;
    if (linha < N && coluna < N)
    {
        float soma = 0.0f;
        for (int i = 0; i < N; i++)
            soma += matrizA[linha * N + i] * matrizB[i * N + coluna];
        matrizC[linha * N + coluna] = soma;
    }
}

int main()
{
    int N;
    std::cin >> N;
    int lengthArray = N * N;
    float *matrizA, *matrizB, *matrizC;
    hipMalloc(&matrizA, lengthArray * sizeof(float));
    hipMalloc(&matrizB, lengthArray * sizeof(float));
    hipMalloc(&matrizC, lengthArray * sizeof(float));
    int threads = N > 32 ? 32 : N;
    int blocks = (N + threads - 1) / threads;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks(blocks, blocks);

    float valor;
    for (int i = 0; i < lengthArray; i++)
    {
        std::cin >> valor;
        hipMemcpy(&matrizA[i], &valor, sizeof(float), hipMemcpyHostToDevice);
    }
    for (int i = 0; i < lengthArray; i++)
    {
        std::cin >> valor;
        hipMemcpy(&matrizB[i], &valor, sizeof(float), hipMemcpyHostToDevice);
    }

    calculaCelulas<<<numBlocks, threadsPerBlock>>>(matrizA, matrizB, matrizC, N);
    for (int i = 0; i < lengthArray; i++)
    {
        hipMemcpy(&valor, &matrizC[i], sizeof(float), hipMemcpyDeviceToHost);
        std::cout << std::fixed << std::setprecision(2) << valor << " ";
        if ((i + 1) % N == 0)
            std::cout << std::endl;
    }
    return 0;
}
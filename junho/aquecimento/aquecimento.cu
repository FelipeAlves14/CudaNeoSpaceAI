#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>

__global__ void calculaCelulas(float *A, float *B, float *produto, int N)
{
    int linha = blockIdx.y * blockDim.y + threadIdx.y;
    int coluna = blockIdx.x * blockDim.x + threadIdx.x;
    if (linha < N && coluna < N)
    {
        float soma = 0.0f;
        for (int i = 0; i < N; i++)
            soma += A[linha * N + i] * B[i * N + coluna];
        produto[linha * N + coluna] = soma;
    }
}

int main()
{
    int N;
    std::cin >> N;
    int lengthArray = N * N;
    float *A, *AHost = new float[lengthArray],
              *B, *BHost = new float[lengthArray],
              *produto, *produtoHost = new float[lengthArray];
    hipMalloc(&A, lengthArray * sizeof(float));
    hipMalloc(&B, lengthArray * sizeof(float));
    hipMalloc(&produto, lengthArray * sizeof(float));

    int threads = N > 32 ? 32 : N;
    int blocks = (N + threads - 1) / threads;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks(blocks, blocks);

    for (int i = 0; i < lengthArray; i++)
        std::cin >> AHost[i];
    hipMemcpy(A, AHost, lengthArray * sizeof(float), hipMemcpyHostToDevice);

    for (int i = 0; i < lengthArray; i++)
        std::cin >> BHost[i];
    hipMemcpy(B, BHost, lengthArray * sizeof(float), hipMemcpyHostToDevice);

    calculaCelulas<<<numBlocks, threadsPerBlock>>>(A, B, produto, N);
    hipDeviceSynchronize();
    hipMemcpy(produtoHost, produto, lengthArray * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < lengthArray; i++)
    {
        std::cout << std::fixed << std::setprecision(2) << produtoHost[i] << " ";
        if ((i + 1) % N == 0)
            std::cout << std::endl;
    }
    hipFree(A);
    hipFree(B);
    hipFree(produto);
    return 0;
}
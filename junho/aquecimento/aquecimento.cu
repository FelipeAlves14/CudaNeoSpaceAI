#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>

__global__ void calculaCelulas(float *matrizA, float *matrizB, float *matrizC, int N)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    float soma = 0.0f;
    for (int i = 0; i < N; i++)
        soma += matrizA[x * N + i] * matrizB[i * N + y];
    matrizC[x * N + y] = soma;
}

int main()
{
    int N;
    std::cin >> N;
    float *matrizA = new float[N * N];
    float *matrizB = new float[N * N];
    float *matrizC = new float[N * N];

    hipMalloc(&matrizA, N * N * sizeof(float));
    hipMalloc(&matrizB, N * N * sizeof(float));
    hipMalloc(&matrizC, N * N * sizeof(float));
    
    dim3 threadsPerBlock(N, N);

    float valor;
    for (int i = 0; i < N * N; i++)
    {
        std::cin >> valor;
        hipMemcpy(&matrizA[i], &valor, sizeof(float), hipMemcpyHostToDevice);
    }
    for (int i = 0; i < N * N; i++)
    {
        std::cin >> valor;
        hipMemcpy(&matrizB[i], &valor, sizeof(float), hipMemcpyHostToDevice);
    }

    calculaCelulas<<<1, threadsPerBlock>>>(matrizA, matrizB, matrizC, N);
    for (int i = 0; i < N * N; i++)
    {
        hipMemcpy(&valor, &matrizC[i], sizeof(float), hipMemcpyDeviceToHost);
        std::cout << std::fixed << std::setprecision(2) << valor << " ";
        if ((i + 1) % N == 0)
            std::cout << std::endl;
    }
    return 0;
}
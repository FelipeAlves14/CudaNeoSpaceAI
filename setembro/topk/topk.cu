#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

__global__ void topk(float *linha_sorted, float *resultado, int N, int topK)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < topK)
        resultado[tid] = linha_sorted[N - topK + tid];
}

int main()
{
    int K, M, N;
    std::cin >> K >> M >> N;

    int length = M * N;
    int topK = std::ceil(N * (K / 100.0));
    int lengthOut = M * topK;

    std::valarray<float> matrizHost(length);
    for (int i = 0; i < length; i++)
        std::cin >> matrizHost[i];

    std::vector<float> outHost(lengthOut);

    for (int row = 0; row < M; row++)
    {
        std::valarray<float> linhaHost = matrizHost[std::slice(row * N, N, 1)];

        float *linha_in, *linha_sorted;
        hipMalloc(&linha_in, N * sizeof(float));
        hipMalloc(&linha_sorted, N * sizeof(float));
        hipMemcpy(linha_in, &linhaHost[0], N * sizeof(float), hipMemcpyHostToDevice);

        void *d_temp_storage = nullptr;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                       linha_in, linha_sorted, N);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
                                       linha_in, linha_sorted, N);

        float *linha_out;
        hipMalloc(&linha_out, topK * sizeof(float));

        int threads = std::min(32, topK);
        int blocks = (topK + threads - 1) / threads;
        topk<<<blocks, threads>>>(linha_sorted, linha_out, N, topK);

        hipMemcpy(&outHost[row * topK], linha_out,
                   topK * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(linha_in);
        hipFree(linha_sorted);
        hipFree(d_temp_storage);
        hipFree(linha_out);
    }

    for (int row = 0; row < M; row++)
    {
        for (int j = topK - 1; j >= 0; j--)
            std::cout << std::fixed << std::setprecision(3) << outHost[row * topK + j] << " ";
        std::cout << std::endl;
    }

    return 0;
}

#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>

__global__ void softmax(float *softmax, float *softmaxTransform, int M, float maxValue)
{
    int linha = blockIdx.x * blockDim.x + threadIdx.x;
    if (linha < M)
        softmaxTransform[linha] = expf(softmax[linha] - maxValue);
}

__global__ void resultSoftmax(float *softmaxTransform, float *result, int M, float sumValue)
{
    int linha = blockIdx.x * blockDim.x + threadIdx.x;
    if (linha < M)
        result[linha] = softmaxTransform[linha] / sumValue;
}

int main()
{
    int M, N;
    std::cin >> M >> N;
    int length = M * N;
    float *matriz, *matrizHost = new float[length];
    hipMalloc(&matriz, length * sizeof(float));
    for (int i = 0; i < length; i++)
        std::cin >> matrizHost[i];
    hipMemcpy(matriz, matrizHost, length * sizeof(float), hipMemcpyHostToDevice);
    while (true)
    {
        int L, C1, C2;
        if (!(std::cin >> L >> C1 >> C2))
            break;

        int softmaxResultLength = C2 - C1 + 1;
        float *softmaxResult, *softmaxResultHost = new float[softmaxResultLength],
                              *maxSoftmaxValue,
                              *softmaxTransform, *softmaxTransformHost = new float[softmaxResultLength],
                              *sumSoftmaxValue;
        float maxSoftmaxValueHost = 0.0f, sumSoftmaxValueHost = 0.0f;
        hipMalloc(&softmaxResult, softmaxResultLength * sizeof(float));
        hipMalloc(&softmaxTransform, softmaxResultLength * sizeof(float));
        hipMalloc(&maxSoftmaxValue, sizeof(float));
        hipMalloc(&sumSoftmaxValue, sizeof(float));

        int linha = L * N;
        for (int i = C1, j = 0; i <= C2; i++, j++)
            softmaxResultHost[j] = matrizHost[linha + i];
        hipMemcpy(softmaxResult, softmaxResultHost, softmaxResultLength * sizeof(float), hipMemcpyHostToDevice);

        void *tempStorage = nullptr;
        size_t tempStorageBytes = 0;
        hipcub::DeviceReduce::Max(tempStorage, tempStorageBytes, softmaxResult, maxSoftmaxValue, softmaxResultLength);
        hipMalloc(&tempStorage, tempStorageBytes);
        hipcub::DeviceReduce::Max(tempStorage, tempStorageBytes, softmaxResult, maxSoftmaxValue, softmaxResultLength);
        hipFree(tempStorage);
        hipMemcpy(&maxSoftmaxValueHost, maxSoftmaxValue, sizeof(float), hipMemcpyDeviceToHost);

        int threads = softmaxResultLength > 32 ? 32 : softmaxResultLength;
        int blocks = (softmaxResultLength + threads - 1) / threads;
        softmax<<<blocks, threads, softmaxResultLength * sizeof(float)>>>(softmaxResult, softmaxTransform, softmaxResultLength, maxSoftmaxValueHost);
        hipDeviceSynchronize();

        void *tempStorage2 = nullptr;
        size_t tempStorageBytes2 = 0;
        hipcub::DeviceReduce::Sum(tempStorage2, tempStorageBytes2, softmaxTransform, sumSoftmaxValue, softmaxResultLength);
        hipMalloc(&tempStorage2, tempStorageBytes2);
        hipcub::DeviceReduce::Sum(tempStorage2, tempStorageBytes2, softmaxTransform, sumSoftmaxValue, softmaxResultLength);
        hipFree(tempStorage2);
        hipMemcpy(&sumSoftmaxValueHost, sumSoftmaxValue, sizeof(float), hipMemcpyDeviceToHost);

        resultSoftmax<<<blocks, threads, softmaxResultLength * sizeof(float)>>>(softmaxTransform, softmaxResult, softmaxResultLength, sumSoftmaxValueHost);
        hipDeviceSynchronize();

        hipMemcpy(softmaxResultHost, softmaxResult, softmaxResultLength * sizeof(float), hipMemcpyDeviceToHost);
        for (int j = 0; j < softmaxResultLength; j++)
        {
            std::cout << std::fixed << std::setprecision(4) << softmaxResultHost[j];
            if (j < C2)
                std::cout << " ";
        }
        std::cout << std::endl;
        hipFree(softmaxResult);
    }
    hipFree(matriz);
    hipFree(softmaxResult);
    hipFree(maxSoftmaxValue);
    hipFree(softmaxTransform);
    hipFree(sumSoftmaxValue);
    return 0;
}
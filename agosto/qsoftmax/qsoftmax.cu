#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>

__global__ void softmax(float *softmax, int M)
{
    int linha = blockIdx.x * blockDim.x + threadIdx.x;
    if (linha < M)
    {
        extern __shared__ float sdata[];

        float max = softmax[linha * M];
        for (int j = 1; j < M; j++)
        {
            float val = softmax[linha * M + j];
            if (val > max)
                max = val;
        }

        float soma = 0.0f;
        for (int j = 0; j < M; j++)
        {
            int index = linha * M + j;
            sdata[index] = expf(softmax[index] - max);
            soma += sdata[index];
        }

        for (int j = 0; j < M; j++)
        {
            int index = linha * M + j;
            softmax[index] = sdata[index] / soma;
        }
    }
}

int main()
{
    int M, N;
    std::cin >> M >> N;
    int length = M * N;
    float *matriz, *matrizHost = new float[length];
    hipMalloc(&matriz, length * sizeof(float));
    for (int i = 0; i < length; i++)
        std::cin >> matrizHost[i];
    hipMemcpy(matriz, matrizHost, length * sizeof(float), hipMemcpyHostToDevice);
    while (true)
    {
        int L, C1, C2;
        if (!(std::cin >> L >> C1 >> C2))
            break;

        int softmaxResultLength = C2 - C1 + 1;
        float *softmaxResult, *softmaxResultHost = new float[softmaxResultLength];
        hipMalloc(&softmaxResult, softmaxResultLength * sizeof(float));

        int linha = L * N;
        for (int i = C1, j = 0; i <= C2; i++, j++)
            softmaxResultHost[j] = matrizHost[linha + i];
        hipMemcpy(softmaxResult, softmaxResultHost, softmaxResultLength * sizeof(float), hipMemcpyHostToDevice);

        int threads = softmaxResultLength > 32 ? 32 : softmaxResultLength;
        int blocks = (softmaxResultLength + threads - 1) / threads;
        softmax<<<blocks, threads, softmaxResultLength * sizeof(float)>>>(softmaxResult, softmaxResultLength);
        hipDeviceSynchronize();
        hipMemcpy(softmaxResultHost, softmaxResult, softmaxResultLength * sizeof(float), hipMemcpyDeviceToHost);
        for (int j = C1; j <= C2; j++)
        {
            std::cout << std::fixed << std::setprecision(4) << softmaxResultHost[L * N + j];
            if (j < C2)
                std::cout << " ";
        }
        std::cout << std::endl;
        hipFree(softmaxResult);
    }
    hipFree(matriz);
    return 0;
}
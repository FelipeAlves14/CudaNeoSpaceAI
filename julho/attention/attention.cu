#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>

__global__ void softmax(float *softmax, int M)
{
    int linha = blockIdx.x * blockDim.x + threadIdx.x;
    if (linha < M)
    {
        extern __shared__ float sdata[];

        float max = softmax[linha * M];
        for (int j = 1; j < M; j++)
        {
            float val = softmax[linha * M + j];
            if (val > max)
                max = val;
        }

        float soma = 0.0f;
        for (int j = 0; j < M; j++)
        {
            int index = linha * M + j;
            sdata[index] = expf(softmax[index] - max);
            soma += sdata[index];
        }

        for (int j = 0; j < M; j++)
        {
            int index = linha * M + j;
            softmax[index] = sdata[index] / soma;
        }
    }
}

__global__ void calculaProduto(float *Q, float *K, float *resultado, int M, int N, bool transposta = false)
{
    int linha = blockIdx.y * blockDim.y + threadIdx.y;
    int coluna = blockIdx.x * blockDim.x + threadIdx.x;
    if (linha < M && coluna < N)
    {
        float soma = 0.0f;
        if (transposta)
        {
            for (int i = 0; i < N; i++)
                soma += Q[linha * N + i] * K[coluna * N + i];
            resultado[linha * M + coluna] = soma / sqrtf(N);
        }
        else
        {
            for (int i = 0; i < N; i++)
                soma += Q[linha * M + i] * K[i * N + coluna];
            resultado[linha * N + coluna] = soma;
        }
    }
}

int main()
{
    int M, N;
    std::cin >> M >> N;
    int arrayLength = M * N;
    int softmaxResultLength = M * M;
    float *Q, *QHost = new float[arrayLength],
              *K, *KHost = new float[arrayLength],
              *V, *VHost = new float[arrayLength],
              *softmaxResult,
              *resultado, *resultadoHost = new float[arrayLength];
    hipMalloc(&Q, arrayLength * sizeof(float));
    hipMalloc(&K, arrayLength * sizeof(float));
    hipMalloc(&V, arrayLength * sizeof(float));
    hipMalloc(&softmaxResult, softmaxResultLength * sizeof(float));
    hipMalloc(&resultado, arrayLength * sizeof(float));

    int threads = M > 32 ? 32 : M;
    int blocks = (M + threads - 1) / threads;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks(blocks, blocks);

    for (int i = 0; i < arrayLength; i++)
        std::cin >> QHost[i];
    hipMemcpy(Q, QHost, arrayLength * sizeof(float), hipMemcpyHostToDevice);

    for (int i = 0; i < arrayLength; i++)
        std::cin >> KHost[i];
    hipMemcpy(K, KHost, arrayLength * sizeof(float), hipMemcpyHostToDevice);

    for (int i = 0; i < arrayLength; i++)
        std::cin >> VHost[i];
    hipMemcpy(V, VHost, arrayLength * sizeof(float), hipMemcpyHostToDevice);

    calculaProduto<<<numBlocks, threadsPerBlock>>>(Q, K, softmaxResult, M, N, true);
    hipDeviceSynchronize();

    softmax<<<blocks, threads, M * sizeof(float)>>>(softmaxResult, M);
    hipDeviceSynchronize();

    threads = N > 32 ? 32 : N;
    threadsPerBlock = dim3(threads, threads);
    blocks = (N + threads - 1) / threads;
    numBlocks = dim3(blocks, blocks);
    calculaProduto<<<numBlocks, threadsPerBlock>>>(softmaxResult, V, resultado, M, N);
    hipDeviceSynchronize();

    hipMemcpy(resultadoHost, resultado, arrayLength * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < arrayLength; i++)
    {
        std::cout << std::fixed << std::setprecision(4) << resultadoHost[i] << " ";
        if ((i + 1) % N == 0)
            std::cout << std::endl;
    }
    hipFree(Q);
    hipFree(K);
    hipFree(V);
    hipFree(softmaxResult);
    hipFree(resultado);
    return 0;
}
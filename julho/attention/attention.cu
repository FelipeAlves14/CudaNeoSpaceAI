#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>

__global__ void calculaProdutoEscalar(float *Q, float *K, float *produtoEscalar, int N, int M)
{
    int linha = blockIdx.y * blockDim.y + threadIdx.y;
    int coluna = blockIdx.x * blockDim.x + threadIdx.x;
    if (linha < N && coluna < M)
    {
        float soma = 0.0f;
        for (int i = 0; i < N; i++)
            soma += Q[linha * N + i] * K[coluna * N + i];
        produtoEscalar[linha * N + coluna] = soma;
    }
}

int main()
{
    int M, N;
    std::cin >> M >> N;
    float *Q, *K, *V, *produto;
    hipMalloc(&Q, M * N * sizeof(float));
    hipMalloc(&K, M * N * sizeof(float));
    hipMalloc(&V, M * N * sizeof(float));
    hipMalloc(&produto, N * N * sizeof(float));

    int threads = M > 32 ? 32 : M;
    int blocks = (M + threads - 1) / threads;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks(blocks, blocks);

    int lengthArray = M * N;
    float valor;

    for (int i = 0; i < lengthArray; i++)
    {
        std::cin >> valor;
        hipMemcpy(&Q[i], &valor, sizeof(float), hipMemcpyHostToDevice);
    }

    for (int i = 0; i < lengthArray; i++)
    {
        std::cin >> valor;
        hipMemcpy(&K[i], &valor, sizeof(float), hipMemcpyHostToDevice);
    }

    for (int i = 0; i < lengthArray; i++)
    {
        std::cin >> valor;
        hipMemcpy(&V[i], &valor, sizeof(float), hipMemcpyHostToDevice);
    }
    calculaProdutoEscalar<<<numBlocks, threadsPerBlock>>>(Q, K, produto, N, M);
    for (int i = 0; i < lengthArray; i++)
    {
        if (!((i + 1) % N == 0))
        {
            hipMemcpy(&valor, &produto[i], sizeof(float), hipMemcpyDeviceToHost);
            std::cout << std::fixed << std::setprecision(4) << valor << " ";
        }
        else
            std::cout << std::endl;
    }
}
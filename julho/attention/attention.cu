#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>

__global__ void softmax(float *produto, int N)
{
    int linha = blockIdx.x * blockDim.x + threadIdx.x;
    if (linha < N)
    {
        extern __shared__ float sdata[];

        float max = produto[linha * N];
        for (int j = 1; j < N; j++)
        {
            float val = produto[linha * N + j];
            if (val > max)
                max = val;
        }

        float soma = 0.0f;
        for (int j = 0; j < N; j++)
        {
            sdata[j] = expf(produto[linha * N + j] - max);
            soma += sdata[j];
        }

        for (int j = 0; j < N; j++)
        {
            produto[linha * N + j] = sdata[j] / soma;
        }
    }
}

__global__ void calculaProduto(float *Q, float *K, float *produto, int N, int M, bool transposta = false)
{
    int linha = blockIdx.y * blockDim.y + threadIdx.y;
    int coluna = blockIdx.x * blockDim.x + threadIdx.x;
    if (linha < N && coluna < M)
    {
        float soma = 0.0f;
        for (int i = 0; i < N; i++)
            soma += Q[linha * N + i] * K[transposta ? coluna * N + i : i * M + coluna];
        produto[linha * N + coluna] = transposta ? soma / sqrtf(N) : soma;
    }
}

int main()
{
    int M, N;
    std::cin >> M >> N;
    int lengthArray = M * N;
    float *Q, *QHost = new float[lengthArray], *K, *KHost = new float[lengthArray], *V, *VHost = new float[lengthArray], *produto, *produtoHost = new float[N * N];
    hipMalloc(&Q, lengthArray * sizeof(float));
    hipMalloc(&K, lengthArray * sizeof(float));
    hipMalloc(&V, lengthArray * sizeof(float));
    hipMalloc(&produto, N * N * sizeof(float));

    int threads = M > 32 ? 32 : M;
    int blocks = (M + threads - 1) / threads;
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks(blocks, blocks);

    for (int i = 0; i < lengthArray; i++)
        std::cin >> QHost[i];
    hipMemcpy(Q, QHost, lengthArray * sizeof(float), hipMemcpyHostToDevice);

    for (int i = 0; i < lengthArray; i++)
        std::cin >> KHost[i];
    hipMemcpy(K, KHost, lengthArray * sizeof(float), hipMemcpyHostToDevice);

    for (int i = 0; i < lengthArray; i++)
        std::cin >> VHost[i];
    hipMemcpy(V, VHost, lengthArray * sizeof(float), hipMemcpyHostToDevice);

    calculaProduto<<<numBlocks, threadsPerBlock>>>(Q, K, produto, N, M, true);
    hipDeviceSynchronize();
    softmax<<<blocks, threads, N * sizeof(float)>>>(produto, M);
    hipDeviceSynchronize();
    hipMemcpy(produtoHost, produto, N * N * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < lengthArray; i++)
    {
        if (!((i + 1) % N == 0))
            std::cout << std::fixed << std::setprecision(4) << produtoHost[i] << " ";
        else
            std::cout << std::endl;
    }
}